
#include <hip/hip_runtime.h>
#include <stdio.h>
#define imin(a,b) (a<b?a:b)
const int N = 33 * 1024;
int main( void ) {
    float   *a, *b, c;
    // allocate memory on the cpu side
    a = (float*)malloc( N*sizeof(float) );
    b = (float*)malloc( N*sizeof(float) );
    // fill in the host memory with data
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i*2;
    }
    // finish up on the CPU side
    c = 0;
    for (int i=0; i<N; i++) {
        c += a[i]*b[i];
    }
    #define sum_squares(x)  (x*(x+1)*(2*x+1)/6)
    printf( "Does CPU value %.6g = %.6g ?\n", c,
             2 * sum_squares( (float)(N - 1) ) );
    // free memory on the cpu side
    free( a );
    free( b );
}
